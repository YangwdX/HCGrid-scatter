#include "hip/hip_runtime.h"
// --------------------------------------------------------------------
//
// title                  :gridding.cu
// description            :Gridding process.
// author                 :
//
// --------------------------------------------------------------------

#include "gridding.h"

/* Initialize output spectrals and weights. */
void init_output(){
    uint32_t num = h_zyx[0] * h_zyx[1] * h_zyx[2];
    h_datacube = RALLOC(double, num);
    h_weightscube = RALLOC(double, num);
    for(uint32_t i = 0; i < num; ++i){
        h_datacube[i] = 0.;
        h_weightscube[i] = 0.;
    }
}

/* Sinc function with simple singularity check. */
double sinc(double x){
    if(fabs(x) < 1.e-10)
        return 1.;
    else
        return sin(x) / x;
}

/* Grid-kernel definitions. get weight*/
double kernel_func_ptr(double distance, double bearing){
    if(h_GMaps.kernel_type == GAUSS1D){   // GAUSS1D
        return exp(-distance * distance * h_kernel_params[0]);
    }
    else if(h_GMaps.kernel_type == GAUSS2D){  // GAUSS2D
        double ellarg = (\
                pow(h_kernel_params[0], 2.0)\
                    * pow(sin(bearing - h_kernel_params[2]), 2.0)\
                + pow(h_kernel_params[1], 2.0)\
                    * pow(cos(bearing - h_kernel_params[2]), 2.0));
        double Earg = pow(distance / h_kernel_params[0] /\
                       h_kernel_params[1], 2.0) / 2. * ellarg;
        return exp(-Earg);
    }
    else if(h_GMaps.kernel_type == TAPERED_SINC){ // TAPERED_SINC
        double arg = PI * distance / h_kernel_params[0];
        return sinc(arg / h_kernel_params[2])\
            * exp(pow(-(arg / h_kernel_params[1]), 2.0));
    }
}

void hcgrid (
        double *h_lons,
        double *h_lats,
        double *h_data,
        double *h_weights,
        double *h_xwcs,
        double *h_ywcs,
        double *h_datacube,
        double *h_weightscube,
        uint64_t *h_hpx_idx) {
        // uint32_t warp_id = blockIdx.x * (blockDim.x / 32) + threadIdx.x / 32;
        // uint32_t tid = ((warp_id % h_GMaps.block_warp_num) * 32 + threadIdx.x % 32) * h_GMaps.factor;
        // printf("\nhere\n"); 
        // printf("%f\n", h_GMaps.sphere_radius);
        uint32_t idx;
        uint32_t xcoord = h_zyx[2];
        uint32_t ycoord = h_zyx[1];
        uint32_t ncoords = h_zyx[1] * h_zyx[2];
        for(idx = 0; idx < h_GMaps.data_shape; idx ++){
            // printf("%d\n", idx);
            double alpha = h_lons[idx] * DEG2RAD;
            double beta = h_lats[idx] * DEG2RAD;
            double in_data = h_data[idx];
            double in_weights = h_weights[idx];
            /* find startpoint*/
            double ubound = h_lats[idx] - h_GMaps.sphere_radius, dbound = h_lats[idx] + h_GMaps.sphere_radius;
            double lbound =  h_lons[idx] + h_GMaps.sphere_radius, rbound = h_lons[idx] - h_GMaps.sphere_radius;
            int lx = 0, rx = 0, uy = 0, dy =0;
            int sy = 0, sp, op;
            /*while(h_xwcs[lx] < lbound && lx < ncoords){
                lx ++;
             }
            lx --;
            while(h_xwcs[rx] < rbound && rx < ncoords){
                rx ++;
            }
            while(h_ywcs[uy] > ubound && uy < ncoords){
                uy += xcoord;
            }
            uy--;
            while(h_ywcs[dy] > dbound && dy < ncoords){
                dy += xcoord;
            }
            for(int xx = 0; xx < xcoord * ycoord; xx ++){
                while(h_xwcs[xx])
            }*/
            while(h_ywcs[sy] < ubound && sy < ncoords){ 
                sy += xcoord;
            }
            if(sy > xcoord){
                sy -= xcoord;
            }
            sp = sy;
            while(h_xwcs[sp] > lbound && sp < ncoords){
                sp ++;
            }
            if(sp > sy){
                sp --;
            }
            /* Gridding*/
            for(int k = sp; h_ywcs[k] < dbound && k < ncoords; k += xcoord){
                for(op = k; h_xwcs[op] > rbound && op < ncoords; op ++){
                    double ga = h_xwcs[op] * DEG2RAD;
                    double  gb = h_ywcs[op] * DEG2RAD;
                    double sdist = true_angular_distance(alpha, beta, ga, gb) * RAD2DEG;
                    double sbear = 0.;
                    if (h_GMaps.bearing_needed) {
                        sbear = great_circle_bearing(alpha, beta, ga, gb);
                    }
                    if(sdist < h_GMaps.sphere_radius){
                        double sweight = kernel_func_ptr(sdist, sbear);
                        double tweight = in_weights * sweight;
                        h_datacube[op] += in_data * tweight;
                        h_weightscube[op] += tweight;
                    }
                }
            }
    }
    return; 

}


/* Gridding process. */
void solve_gridding(const char *infile, const char *tarfile, const char *outfile, const char *sortfile, const int& param, const int &bDim) {
    double iTime1 = cpuSecond();
    // Read input points.
    //reah_input_map_hdf5(infile);
    // printf("\nhere\n");
    read_input_map(infile);
    // Read output map.
    read_output_map(tarfile);

    // Set wcs for output pixels.
    set_WCS();

    // Initialize output spectrals and weights.
    init_output();

//    iTime2 = cpuSecond();
    // Block Indirect Sort i nput points by their healpix indexes.
    // if (param == THRUST) { 
    //     init_input_with_thrust(param);
    // } else {
    //     init_input_with_cpu(param);
    // }

    double iTime3 = cpuSecond();
    // Alloc data for GPU.
    // data_alloc();

    double iTime4 = cpuSecond();
    // Send data from CPU to GPU.
    // data_h2d();
    printf("h_zyx[1]=%d, h_zyx[2]=%d, ", h_zyx[1], h_zyx[2]);
    // for(int i = 0; i < h_zyx[1]; i++){
    //     for(int j = 0; j < h_zyx[2]; j++){
    //         printf("%f ", h_ywcs[i*90 + j]);
    //     }
    //     printf("\n");
    // }

    // Set block and thread.
    // dim3 block(bDim);
    // dim3 grid((h_GMaps.block_warp_num * h_zyx[1] - 1) / (block.x / 32) + 1);
    // printf("grid.x=%d, block.x=%d, ", grid.x, block.x);

    // Get start time.
    // hipEvent_t start, stop;
    // HANDLE_ERROR(hipEventCreate(&start));
    // HANDLE_ERROR(hipEventCreate(&stop));
    // HANDLE_ERROR(hipEventRecord(start, 0));
    
    hcgrid(h_lons, h_lats, h_data, h_weights, h_xwcs, h_ywcs, h_datacube, h_weightscube, h_hpx_idx);
    
    // Get stop time.
    // printf("kernel elapsed time=%f, ", elapsedTime);

    // Send data from GPU to CPU
    // data_d2h();

    // Write output FITS file
    write_output_map(outfile);

    // Write sorted input FITS file
    if (sortfile) {
        write_ordered_map(infile, sortfile);
    }

    // Release data
    // data_free();
    // HANDLE_ERROR( hipEventDestroy(start) );
    // HANDLE_ERROR( hipEventDestroy(stop) );
    // HANDLE_ERROR( hipDeviceReset() );

    double iTime5 = cpuSecond();
    double iElaps = (iTime5 - iTime1) * 1000.;
    printf("solving_gridding time=%f\n", iElaps);
}